#include "BlackHole.h"
#include ""

#define G 6.67430e-11f
#define EPSILON 1e-6f

__host__ __device__ float BlackHole::getGravity(float r) const {
    return G * mass / ((r + EPSILON) * (r + EPSILON));
}
