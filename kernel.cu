#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <iostream>
#include <cstdlib>

using uint8_t = unsigned char;

struct Vec3 {
    float x, y, z;

    __device__ Vec3 operator+(Vec3 other) const {
        return { x + other.x, y + other.y, z + other.z };
    }

    __device__ Vec3 operator-(Vec3 other) const {
        return { x - other.x, y - other.y, z - other.z };
    }

    __device__ Vec3 operator*(float d) const {
        return { x * d, y * d, z * d };
    }
};

struct Photon {
    Vec3 position;
    Vec3 velocity;

    __device__ void update(float dt, Vec3 blackHolePos, float blackHoleMass) {
        Vec3 direction = blackHolePos - position;
        float distSq = direction.x * direction.x + direction.y * direction.y + direction.z * direction.z;
        float force = blackHoleMass / (distSq + 0.1f); // Unikamy dzielenia przez 0
        velocity = velocity + direction * force * dt;
        position = position + velocity * dt;
    }
};

__constant__ Vec3 blackHolePos;
__constant__ float blackHoleMass;

Photon* d_photons;
uint8_t* colorField;
size_t numPhotons = 2000;
size_t fieldWidth, fieldHeight;

#define CUDA_CALL(x) hipError_t error = hipGetLastError(); if (error != hipSuccess) { std::cout << hipGetErrorName(error) << std::endl; std::abort(); } x

void cudaInit(size_t width, size_t height) {
    fieldWidth = width;
    fieldHeight = height;
    hipMalloc(&d_photons, numPhotons * sizeof(Photon));
    hipMalloc(&colorField, fieldWidth * fieldHeight * 4 * sizeof(uint8_t));

    // Ustawienie stałej pozycji czarnej dziury
    Vec3 h_blackHolePos = { fieldWidth / 2.0f, fieldHeight / 2.0f, 0.0f };
    float h_blackHoleMass = 1.0e+3f;
    hipMemcpyToSymbol(HIP_SYMBOL(blackHolePos), &h_blackHolePos, sizeof(Vec3));
    hipMemcpyToSymbol(HIP_SYMBOL(blackHoleMass), &h_blackHoleMass, sizeof(float));

    // Inicjalizacja fotonów
    Photon* h_photons = new Photon[numPhotons];
    for (size_t i = 0; i < numPhotons; i++) {
        float angle = (2.0f * 3.14159265359f * i) / numPhotons;
        h_photons[i].position = { fieldWidth / 2.0f, fieldHeight / 2.0f -200.0f, 0.0f };
        h_photons[i].velocity = { cos(angle) * 10.0f, sin(angle) * 10.0f, 0.0f };
    }
    hipMemcpy(d_photons, h_photons, numPhotons * sizeof(Photon), hipMemcpyHostToDevice);
    delete[] h_photons;
}

__global__ void updatePhotons(Photon* photons, size_t numPhotons, float dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numPhotons) return;
    if (i == 0) {
        printf("Photon 0: x=%f, y=%f, z=%f\n", photons[i].position.x, photons[i].position.y, photons[i].position.z);
    }
    photons[i].update(dt, blackHolePos, blackHoleMass);
}

__global__ void renderPhotons(uint8_t* colorField, Photon* photons, size_t numPhotons, size_t width, size_t height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numPhotons) return;

    int x = (int)photons[i].position.x;
    int y = (int)photons[i].position.y;
    if (x >= 0 && x < width && y >= 0 && y < height) {
        int idx = (y * width + x) * 4;
        colorField[idx] = 255;     // Czerwony
        colorField[idx + 1] = 0;   // Zielony (zero)
        colorField[idx + 2] = 0;   // Niebieski (zero)
        colorField[idx + 3] = 255; // Alpha (pełna widoczność)
    }

}

void computeField(uint8_t* result, float dt) {
    dim3 threadsPerBlock(256);
    dim3 numBlocks((numPhotons + threadsPerBlock.x - 1) / threadsPerBlock.x);
    updatePhotons << <numBlocks, threadsPerBlock >> > (d_photons, numPhotons, dt);
    hipMemset(colorField, 50, fieldWidth * fieldHeight * 4 * sizeof(uint8_t)); // Ustawienie tła na szaro

    renderPhotons << <numBlocks, threadsPerBlock >> > (colorField, d_photons, numPhotons, fieldWidth, fieldHeight);
    hipMemcpy(result, colorField, fieldWidth * fieldHeight * 4 * sizeof(uint8_t), hipMemcpyDeviceToHost);
}

void cudaExit() {
    hipFree(d_photons);
    hipFree(colorField);
}
